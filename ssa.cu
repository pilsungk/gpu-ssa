#include "hip/hip_runtime.h"
/**
 *  FILE:    SSA.cu
 *  AUTHOR:  Pilsung Kang
 *  CREATED: July 16, 2008
 *  LAST MODIFIED: Aug 14, 2017 
 *             BY: Pilsung Kang
 *             TO: make it work on CUDA 8 on GTX 1080ti 
 *
 *  SUMMARY:
 *
 *  NOTES: Adapted from StochKit
 *  TO DO: Making it work on CUDA 8
 *
 * cutil.h was removed:
 * - CUDA_SAFE_CALL ==> checkCudaErrors
 */
 
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

// Helper functions and utilities to work with CUDA
// Ned to specify the "CUDA_Sample_dir/common/inc" dir when compiling
#include <helper_functions.h>
#include <hip/hip_runtime_api.h> 

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


/* problem parameters and cuda threads launch geometry */
#include "prob_params.h"

__global__ void ssa_kernel(hiprandState_t* states, int *x, float *ftime);

static const int x[NX] = {1200, 600, 0};

static void init_x_array(int *xarr)
{
    for (int i=0; i<NTHREADS; i++)
        for (int j=0; j<NX; j++) 
            xarr[NX*i+j] = x[j];
}

/* this GPU kernel function is used to initialize the random states */
__global__ void init(unsigned int seed, hiprandState_t* states) {
    /* we have to initialize the state */
    hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
                blockIdx.x*blockDim.x+threadIdx.x, /* the sequence number should be different for each core (unless you want all
                           cores to get the same sequence of numbers for some reason - use thread id! */
                0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                &states[blockIdx.x*blockDim.x+threadIdx.x]);
}

/* this GPU kernel takes an array of states, and an array of ints, and puts a
 * random float between 0.0 and 1.0 into each */
__global__ void randoms(hiprandState_t* states, float* numbers) {
    /* hiprand works like rand - except that it takes a state as a parameter */
	numbers[blockIdx.x*blockDim.x+threadIdx.x] = hiprand_uniform(&states[blockIdx.x*blockDim.x+threadIdx.x]);
}


int main(int argc, char** argv) {

    // In my Linux config with 1080ti, the id is 0
    int devID = 0;

    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&devID);

    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    hiprandState_t* states;

    /* allocate space on the GPU for the random states */
    hipMalloc((void**) &states, NTHREADS * sizeof(hiprandState_t));

    dim3 dimBlock(XBLOCKSIZE, YBLOCKSIZE);
    dim3 dimGrid(XGRIDSIZE, YGRIDSIZE);

    /* invoke the GPU to initialize all of the random states */
    init<<<dimGrid, dimBlock>>>(time(0), states);

	/* allocate an array of floats on the CPU and GPU */
	float cpu_nums[NTHREADS];
	float* gpu_nums;
	hipMalloc((void**) &gpu_nums, NTHREADS * sizeof(float));

	/* invoke the kernel to get some random numbers */
	randoms<<<dimGrid, dimBlock>>>(states, gpu_nums);

	/* copy the random numbers back */
	hipMemcpy(cpu_nums, gpu_nums, NTHREADS * sizeof(float), hipMemcpyDeviceToHost);

	/* print them out */
	// for (int i = 0; i < NTHREADS; i++) {
	// 	printf("%10.10f\n", cpu_nums[i]);
	// }


    // allocate dev mem for x (specifes) and copy initial values
    int *x_array = (int *) malloc(NX*NTHREADS*sizeof(int));
    init_x_array(x_array);
    int *dev_x_array;

    checkCudaErrors(hipMalloc((void **)&dev_x_array, sizeof(int)*NX*NTHREADS));
    checkCudaErrors(hipMemcpy(dev_x_array, x_array, sizeof(int)*NX*NTHREADS, hipMemcpyHostToDevice));

    // allocate dev mem for final time and copy initial values
    float *finalT_array = (float *) malloc(NTHREADS*sizeof(float));
    float *dev_finalT_array;

    checkCudaErrors(hipMalloc((void **)&dev_finalT_array, sizeof(float)*NTHREADS));

    // Referenced from the matrixMul CUDA sample
    // Allocate CUDA events that we'll use for timing
    hipEvent_t start;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Record the start event
    error = hipEventRecord(start, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    printf("Number of CUDA threads = %u \n", NTHREADS);

    // Execute the SSA kernel
    ssa_kernel<<<dimGrid, dimBlock>>>(states, dev_x_array, dev_finalT_array);
    // device sync might be unnecessary...
	hipDeviceSynchronize();  

    // Record the stop event
    error = hipEventRecord(stop, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    printf("Kernel exec time = %.3f msec\n", msecTotal);

    // copy result from device to host
    checkCudaErrors(hipMemcpy(x_array, dev_x_array, sizeof(int)*NX*NTHREADS, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(finalT_array, dev_finalT_array, sizeof(float)*NTHREADS, hipMemcpyDeviceToHost));

    // for (int i=0; i<NTHREADS; i++) {
    //     printf("Tid %d at final time %f: %d\t\t%d\t\t%d\n",
    //         i, finalT_array[i], x_array[i*NX], x_array[i*NX+1], x_array[i*NX+2]);
    // }

	/* free the memory we allocated for the states and numbers */
    free(x_array); 
    free(finalT_array);
	checkCudaErrors(hipFree(states));
	checkCudaErrors(hipFree(gpu_nums));
    checkCudaErrors(hipFree(dev_x_array)); 
    checkCudaErrors(hipFree(dev_finalT_array));

    return 0;
}

